#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <time.h>

#include "utils.h"

__global__ void matrixMultiplyKernel(double *A, double *B, double *C, int rowsA, int colsA, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB) {
        for (int k = 0; k < colsA; ++k) {
            C[row * colsB + col] += A[row * colsA + k] * B[k * colsB + col];
        }
    }
}

double *matrix_multiply_cuda(double *A, double *B, int rowsA, int colsA, int colsB) {
    double *C = (double *)malloc(rowsA * colsB * sizeof(double));

    size_t sizeA = rowsA * colsA * sizeof(double);
    size_t sizeB = colsA * colsB * sizeof(double);
    size_t sizeC = rowsA * colsB * sizeof(double);

    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, sizeA);
    hipMalloc((void **)&d_B, sizeB);
    hipMalloc((void **)&d_C, sizeC);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((colsB + blockDim.x - 1) / blockDim.x, (rowsA + blockDim.y - 1) / blockDim.y);

    matrixMultiplyKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, rowsA, colsA, colsB);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return C;
}

int main(int argc, char *argv[]) {
    if (argc != 4) {
        fprintf(stderr, "Usage: %s <matrixA.txt> <matrixB.txt> <result.txt>\n", argv[0]);
        return -1;
    }

    int rowsA, colsA, rowsB, colsB;
    double *A, *B, *C;

    A = read_double_matrix(argv[1], &rowsA, &colsA);
    B = read_double_matrix(argv[2], &rowsB, &colsB);

    validate_dimensions(rowsA, colsA, rowsB, colsB);

    printf("Done loading data, starting computations\n");

    double start_time = clock();
    C = matrix_multiply_cuda(A, B, rowsA, colsA, colsB);
    double end_time = clock();

    printf("Matrix multiplication completed in %lf seconds\n", (double)(end_time - start_time) / CLOCKS_PER_SEC);

    write_double_matrix(argv[3], C, rowsA, colsB);

    // Free allocated memory
    free(A);
    free(B);
    free(C);

    return 0;
}