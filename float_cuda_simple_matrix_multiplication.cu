#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

float *read_matrix(const char *filename, int *rows, int *cols) {
    FILE *file = fopen(filename, "r");
    if (!file) {
        fprintf(stderr, "Error opening file\n");
        exit(-1);
    }

    if (fscanf(file, "%d %d\n", rows, cols) != 2) {
        fprintf(stderr, "Invalid matrix format\n");
        fclose(file);
        exit(-1);
    }

    float *matrix = (float *)malloc((*rows) * (*cols) * sizeof(float));

    for (int i = 0; i < (*rows) * (*cols); i++) {
        if (fscanf(file, "%f", &matrix[i]) != 1) {
            fprintf(stderr, "Invalid matrix data\n");
            fclose(file);
            exit(-1);
        }
    }

    fclose(file);
    return matrix;
}

void write_matrix(const char *filename, float *matrix, int rows, int cols) {
    FILE *file = fopen(filename, "w");
    if (!file) {
        fprintf(stderr, "Error opening file\n");
        exit(-1);
    }

    fprintf(file, "%d %d\n", rows, cols);

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            fprintf(file, "%f ", matrix[i * cols + j]);
        }
        fprintf(file, "\n");
    }

    fclose(file);
}

void validate_dimensions(int rowsA, int colsA, int rowsB, int colsB) {
    if (colsA != rowsB) {
        fprintf(stderr, "Matrix dimensions mismatch: %d != %d\n", colsA, rowsB);
        exit(-1);
    }
}

__global__ void matrixMultiplyKernel(float *A, float *B, float *C, int rowsA, int colsA, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB) {
        for (int k = 0; k < colsA; ++k) {
            C[row * colsB + col] += A[row * colsA + k] * B[k * colsB + col];
        }
    }
}

float *matrix_multiply_cuda(float *A, float *B, int rowsA, int colsA, int colsB) {
    float *C = (float *)malloc(rowsA * colsB * sizeof(float));

    size_t sizeA = rowsA * colsA * sizeof(float);
    size_t sizeB = colsA * colsB * sizeof(float);
    size_t sizeC = rowsA * colsB * sizeof(float);

    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, sizeA);
    hipMalloc((void **)&d_B, sizeB);
    hipMalloc((void **)&d_C, sizeC);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((colsB + blockDim.x - 1) / blockDim.x, (rowsA + blockDim.y - 1) / blockDim.y);

    matrixMultiplyKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, rowsA, colsA, colsB);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return C;
}

int main(int argc, char *argv[]) {
    if (argc != 4) {
        fprintf(stderr, "Usage: %s <matrixA.txt> <matrixB.txt> <result.txt>\n", argv[0]);
        return -1;
    }

    int rowsA, colsA, rowsB, colsB;
    float *A, *B, *C;

    A = read_matrix(argv[1], &rowsA, &colsA);
    B = read_matrix(argv[2], &rowsB, &colsB);

    validate_dimensions(rowsA, colsA, rowsB, colsB);

    printf("Done loading data, starting computations\n");

    float start_time = clock();
    C = matrix_multiply_cuda(A, B, rowsA, colsA, colsB);
    float end_time = clock();

    printf("Matrix multiplication completed in %f seconds\n", (float)(end_time - start_time) / CLOCKS_PER_SEC);

    write_matrix(argv[3], C, rowsA, colsB);

    // Free allocated memory
    free(A);
    free(B);
    free(C);

    return 0;
}