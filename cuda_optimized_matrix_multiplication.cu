#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define BLOCK_SIZE 4
#define SHARED_CACHE_PER_THREAD_SIZE 2
#define SHARED_CACHE_SIZE SHARED_CACHE_PER_THREAD_SIZE * BLOCK_SIZE
// Divide x/y and round up
#define CEIL_DIVISION(x, y) ((x) + (y) - 1)/(y)

double *read_matrix(const char *filename, int *rows, int *cols) {
    FILE *file = fopen(filename, "r");
    if (!file) {
        fprintf(stderr, "Error opening file\n");
        exit(-1);
    }

    if (fscanf(file, "%d %d\n", rows, cols) != 2) {
        fprintf(stderr, "Invalid matrix format\n");
        fclose(file);
        exit(-1);
    }

    double *matrix = (double *)malloc((*rows) * (*cols) * sizeof(double));

    for (int i = 0; i < (*rows) * (*cols); i++) {
        if (fscanf(file, "%lf", &matrix[i]) != 1) {
            fprintf(stderr, "Invalid matrix data\n");
            fclose(file);
            exit(-1);
        }
    }

    fclose(file);
    return matrix;
}

void write_matrix(const char *filename, double *matrix, int rows, int cols) {
    FILE *file = fopen(filename, "w");
    if (!file) {
        fprintf(stderr, "Error opening file\n");
        exit(-1);
    }

    fprintf(file, "%d %d\n", rows, cols);

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            fprintf(file, "%lf ", matrix[i * cols + j]);
        }
        fprintf(file, "\n");
    }

    fclose(file);
}

void validate_dimensions(int rowsA, int colsA, int rowsB, int colsB) {
    if (colsA != rowsB) {
        fprintf(stderr, "Matrix dimensions mismatch: %d != %d\n", colsA, rowsB);
        exit(-1);
    }
}

__global__ void matrixMultiplyKernel(double *A, double *B, double *C, int rowsA, int colsA, int colsB) {
    int startSubMatrixRow = threadIdx.y * SHARED_CACHE_PER_THREAD_SIZE;
    int startSubMatrixCol = threadIdx.x * SHARED_CACHE_PER_THREAD_SIZE;

    // Row along which given thread moves through A and column along which it moves through B are constant 
    int startRowA = blockIdx.y * blockDim.y * SHARED_CACHE_PER_THREAD_SIZE + startSubMatrixRow;
    int startColB = blockIdx.x * blockDim.x * SHARED_CACHE_PER_THREAD_SIZE + startSubMatrixCol;

    __shared__ double As [SHARED_CACHE_SIZE][SHARED_CACHE_SIZE];
    __shared__ double Bs [SHARED_CACHE_SIZE][SHARED_CACHE_SIZE];

    double acc[SHARED_CACHE_PER_THREAD_SIZE][SHARED_CACHE_PER_THREAD_SIZE] = {0};
    double a_vals[SHARED_CACHE_PER_THREAD_SIZE];
    double b_vals[SHARED_CACHE_PER_THREAD_SIZE];

    int currentRowA, currentColA, currentRowB, currentColB;


    // Each block loads a chunk of A and B matrices into shared memory
    // Each thread loads SHARED_CACHE_PER_THREAD_SIZE by SHARED_CACHE_PER_THREAD_SIZE subchunk
    // So entire chunk is a size of SHARED_CACHE_SIZE (BLOCK_SIZE * SHARED_CACHE_PER_THREAD_SIZE)
    for (int blockOffset = 0; blockOffset < colsA; blockOffset+=SHARED_CACHE_SIZE) { 
        // Padding the matrices and keeping the kernel without range checking actually makes it slower
        
        // Load all elements from A to As
        #pragma unroll
        for (int loadRowOffset = 0; loadRowOffset < SHARED_CACHE_PER_THREAD_SIZE; loadRowOffset++) {
            #pragma unroll
            for (int loadColOffset = 0; loadColOffset < SHARED_CACHE_PER_THREAD_SIZE; loadColOffset++) {
                currentRowA = startRowA + loadRowOffset;
                currentColA = blockOffset + startSubMatrixCol + loadColOffset;
                if (currentRowA < rowsA && currentColA < colsA) {
                    As[startSubMatrixRow + loadRowOffset][startSubMatrixCol + loadColOffset] = A[currentRowA * colsA + currentColA];
                }
                else {
                    As[startSubMatrixRow + loadRowOffset][startSubMatrixCol + loadColOffset] = 0.0;
                }
            }
        }

        // Load all elements from B to Bs
        #pragma unroll
        for (int loadRowOffset = 0; loadRowOffset < SHARED_CACHE_PER_THREAD_SIZE; loadRowOffset++) {
            #pragma unroll
            for (int loadColOffset = 0; loadColOffset < SHARED_CACHE_PER_THREAD_SIZE; loadColOffset++) {
                currentRowB = blockOffset + startSubMatrixRow + loadRowOffset;
                currentColB = startColB + loadColOffset;
                if (currentRowA < colsA && currentColB < colsB) {
                    Bs[startSubMatrixRow + loadRowOffset][startSubMatrixCol + loadColOffset] = A[currentRowB * colsB + currentColB];
                }
                else {
                    Bs[startSubMatrixRow + loadRowOffset][startSubMatrixCol + loadColOffset] = 0.0;
                }
            }
        }

        // Perform multiplication and accumulation
        #pragma unroll
        for (int sharedTileIndex = 0; sharedTileIndex < SHARED_CACHE_SIZE; ++sharedTileIndex) {

            // Load values from shared memory into registers
            #pragma unroll
            for (int subRow = 0; subRow < SHARED_CACHE_PER_THREAD_SIZE; ++subRow) {
                a_vals[subRow] = As[startSubMatrixRow + subRow][sharedTileIndex];
            }

            #pragma unroll
            for (int subCol = 0; subCol < SHARED_CACHE_PER_THREAD_SIZE; ++subCol) {
                b_vals[subCol] = Bs[sharedTileIndex][startSubMatrixCol + subCol];
            }

            // Compute products and accumulate in registers
            #pragma unroll
            for (int subRow = 0; subRow < SHARED_CACHE_PER_THREAD_SIZE; ++subRow) {
                #pragma unroll
                for (int subCol = 0; subCol < SHARED_CACHE_PER_THREAD_SIZE; ++subCol) {
                    acc[subRow][subCol] += a_vals[subRow] * b_vals[subCol];
                }
            }
        }
        __syncthreads();
    }


    // Write the accumulated values back to C
    #pragma unroll
    for (int subRow = 0; subRow < SHARED_CACHE_PER_THREAD_SIZE; ++subRow) {
        int globalRow = startRowA + subRow;
        printf("%d\n", globalRow);
        if (globalRow >= rowsA) continue;
        #pragma unroll
        for (int subCol = 0; subCol < SHARED_CACHE_PER_THREAD_SIZE; ++subCol) {
            int globalCol = startColB + subCol;
            if (globalCol >= colsB) continue;
            printf("%d %d\n", globalRow, globalCol);
            if (globalRow < 10 && globalCol < 10) {
                printf("%lf\n", acc[subRow][subCol]);
            }
            C[globalRow * colsB + globalCol] = acc[subRow][subCol];
        }
    }
}

double *matrix_multiply_cuda(double *A, double *B, int rowsA, int colsA, int colsB) {
    double *C = (double *)malloc(rowsA * colsB * sizeof(double));

    size_t sizeA = rowsA * colsA * sizeof(double);
    size_t sizeB = colsA * colsB * sizeof(double);
    size_t sizeC = rowsA * colsB * sizeof(double);

    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, sizeA);
    hipMalloc((void **)&d_B, sizeB);
    hipMalloc((void **)&d_C, sizeC);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim(CEIL_DIVISION(colsB, blockDim.x), CEIL_DIVISION(rowsA, blockDim.y));

    matrixMultiplyKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, rowsA, colsA, colsB);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return C;
}

int main(int argc, char *argv[]) {
    if (argc != 4) {
        fprintf(stderr, "Usage: %s <matrixA.txt> <matrixB.txt> <result.txt>\n", argv[0]);
        return -1;
    }

    int rowsA, colsA, rowsB, colsB;
    double *A, *B, *C;

    A = read_matrix(argv[1], &rowsA, &colsA);
    B = read_matrix(argv[2], &rowsB, &colsB);

    validate_dimensions(rowsA, colsA, rowsB, colsB);

    printf("Done loading data, starting computations\n");

    double start_time = clock();
    C = matrix_multiply_cuda(A, B, rowsA, colsA, colsB);
    double end_time = clock();

    printf("Matrix multiplication completed in %f seconds\n", (double)(end_time - start_time) / CLOCKS_PER_SEC);

    write_matrix(argv[3], C, rowsA, colsB);

    // Free allocated memory
    free(A);
    free(B);
    free(C);

    return 0;
}